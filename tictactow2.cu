
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define X 1
#define O -1
#define BLANK 0

#define NUM_THREAD 500
#define RUNS_PER_THREAD 20
#define NUM_BLOCKS 20
#define NUM_SEQ_LOOPS 200000

//When a subBoard is won, it should be filled with that mark. (All X or all O) This is needed as a speed optimization

__device__ __host__ void printSquare(int x){
    switch(x)
    {
        case X:
            printf("X");
            break;
        case O:
            printf("O");
            break;
        case BLANK:
            printf("_");
            break;
    }
}

bool printSquareWithNumber(int x, int num){
    switch(x)
    {
        case X:
            printf("XX");
            return false;
        case O:
            printf("OO");
            return false;
        case BLANK:
            printf("%d", num);
            if(num<=9) { printf(" ");}
            return true;
    }
    return false;
}

void PrintBoardWithNumbers(int* board){
    int x,y,j,i, num;
    num = 0;
    for(x = 0; x <3; x++)
    {
        printf("_____________\n");
        for(i = 0; i < 3; i++)
        {
            printf("|");
            for(y = 0; y < 3; y++)
            {
                for(j = 0; j< 3; j++)
                {
                    if(printSquareWithNumber(board[(3*x+y)*9+(3*i+j)],num))
                    {
                        num++;
                    }
                }
                printf("|");
            }
            printf("\n");
        }
    }
    printf("_____________\n");
}

void PrintBoard(int* board){
    int x,y,j,i;
    for(x = 0; x <3; x++)
    {
        printf("_____________\n");
        for(i = 0; i < 3; i++)
        {
            printf("|");
            for(y = 0; y < 3; y++)
            {
                for(j = 0; j< 3; j++)
                {
                    printSquare(board[(3*x+y)*9+(3*i+j)]);
                }
                printf("|");
            }
            printf("\n");
        }
    }
    printf("_____________\n");
}

void PrintSubBoardWithNumbers(int* subBoard){
    int x,y, num;
    num = 0;
    for(x = 0; x < 3; x++)
    {
        printf("|");
        for(y = 0; y < 3; y++)
        {
            if(printSquareWithNumber(subBoard[3*x+y],num))
            {
                num++;
            }
        }
        printf("|\n");
    }
}

void PrintSubBoard(int* subBoard){
    int x,y;
    for(x = 0; x < 3; x++)
    {
        printf("|");
        for(y = 0; y < 3; y++)
        {
            printSquare(subBoard[3*x+y]);
        }
        printf("|\n");
    }
}

__device__ __host__ int SubBoardWinner(int* subBoard){
    int i, total;
    //left to right wins
    for(i = 0; i < 3; i++)
    {
        total = subBoard[3*i] + subBoard[3*i +1 ] + subBoard[3*i+2];
       // printf("total: %d\n",total);
        if(abs(total) == 3)
        {
            return (total/3);
        }
    }
    //up to down
    for(i = 0; i < 3; i++)
    {
        total = subBoard[i] + subBoard[3+i] + subBoard[6+i];
       // printf("total: %d\n",total);
        if(abs(total) == 3)
        {
            return (total/3);
        }
    }
    //Diagonals
    total = subBoard[0] + subBoard[4] + subBoard[8];
    //printf("total: %d\n",total);
    if(abs(total) == 3)
    {
        return (total/3);
    }
    total = subBoard[2] + subBoard[4] + subBoard[6];
   // printf("total: %d\n",total);
    if(abs(total) == 3)
    {
        return (total/3);
    }
    return 0;
}

int SubBoardWinner(double* subBoard){
    int i, total;
    //left to right wins
    for(i = 0; i < 3; i++)
    {
        total = subBoard[3*i] + subBoard[3*i +1 ] + subBoard[3*i+2];
        if(abs(total) == 3)
        {
            return (total/3);
        }
    }
    //up to down
    for(i = 0; i < 3; i++)
    {
        total = subBoard[i] + subBoard[  3+i ] + subBoard[6+i];
        if(abs(total) == 3)
        {
            return (total/3);
        }
    }
    //Diagonals
    total = subBoard[0] + subBoard[4] + subBoard[8];
    if(abs(total) == 3)
    {
        return (total/3);
    }
    total = subBoard[2] + subBoard[4] + subBoard[6];
    if(abs(total) == 3)
    {
        return (total/3);
    }
    return 0;
}

__device__ __host__ int BoardWinner(int* board){
    int i,metaBoard[9];
    for(i = 0; i < 9; i++)
    {
        metaBoard[i] = SubBoardWinner(board+(i*9));
    }
    return SubBoardWinner(metaBoard);
}

__device__ __host__ bool IsSubBoardFull(int* subBoard){
    int i;
    for(i = 0; i < 9; i++)
    {
        if(subBoard[i] == 0)
        {
            return false;
        }
    }
    return true;
}

__device__ __host__ bool IsSubBoardFull(double* subBoard){
    int i;
    for(i = 0; i < 9; i++)
    {
        if(subBoard[i] != 1 || subBoard[i] != -1)
        {
            return false;
        }
    }
    return true;
}

__device__ __host__ bool IsBoardFull(int* board){
    for(int i = 0; i < 81; i++)
    {
        if(board[i] == BLANK)
        {
            return false;
        }
    }
    return true;
}

__device__ __host__ int NumberOfFreeSquaresInFullBoard(int* board){
    int i, count = 0;
    for(i = 0; i < 81; i++)
    {
        if(board[i] == 0)
        {
            count++;
        }
    }
    return count;
}

__device__ __host__ int NumberOfFreeSquaresInSubBoard(int* subBoard){
    int i, count = 0;
    for(i = 0; i < 9; i++)
    {
        if(subBoard[i] == 0)
        {
            count++;
        }
    }
    return count;
}

__device__ __host__ int NumberOfPossibleMoves(int* board, int lastMove, bool fullBoard){
    int subBoard = lastMove % 9;
    if(fullBoard)
    {
        return NumberOfFreeSquaresInFullBoard(board);
    }
    else
    {
        return NumberOfFreeSquaresInSubBoard(board + 9*subBoard);
    }
}

int DoEvalRow(int a, int b, int c){
    int count = 0;
    int sum = a + b + c;
    if(a != 0) {count++;}
    if(b != 0) {count++;}
    if(c != 0) {count++;}
    return sum * count;
}

double DoEvalRow(double a, double b, double c){
    int count = 0;
    double sum = a + b + c;
    if(sum > 0)
    {
        if(a > 0) {count++;}
        if(b > 0) {count++;}
        if(c > 0) {count++;}
    }
    else
    {
        if(a < 0) {count++;}
        if(b < 0) {count++;}
        if(c < 0) {count++;}
    }
    return sum * (double)count;
}

int EvalRow(int a, int b, int c){
    if( a >= 0 && b >= 0  && c >= 0)
    {
        return DoEvalRow(a,b,c);
    }
    else if( a <= 0 && b <= 0 && c <= 0)
    {
        return DoEvalRow(a,b,c);
    }
    else
    {
        return 0;
    }
}

double EvalSubBoard(int* subBoard){
    double sum = 0;
    int winner = SubBoardWinner(subBoard);
    switch (winner)
    {
        case BLANK:
            if(IsSubBoardFull(subBoard))
            {
                return 0;
            }
            sum += EvalRow(subBoard[0],subBoard[1],subBoard[2]);
            sum += EvalRow(subBoard[3],subBoard[4],subBoard[5]);
            sum += EvalRow(subBoard[6],subBoard[7],subBoard[8]);
            sum += EvalRow(subBoard[0],subBoard[3],subBoard[6]);
            sum += EvalRow(subBoard[1],subBoard[4],subBoard[7]);
            sum += EvalRow(subBoard[2],subBoard[5],subBoard[8]);
            sum += EvalRow(subBoard[0],subBoard[4],subBoard[8]);
            sum += EvalRow(subBoard[2],subBoard[4],subBoard[6]);
            sum /= 21;
            break;
        case  X:
            sum = 1;
            break;
        case O:
            sum = -1;
            break;
    }
    return sum;
}

double EvalMetaRow(double a, double b, double c){
    if( a > -1 && b > -1  && c > -1)
    {
        return DoEvalRow(a,b,c);
    }
    else if( a < 1 && b < 1 && c < 1)
    {
        return DoEvalRow(a,b,c);
    }
    else
    {
        return 0;
    }
}

double EvalMetaBoard(double* subBoard){
    double sum = 0;
    int winner = SubBoardWinner(subBoard);
    switch (winner)
    {
        case BLANK:
            if(IsSubBoardFull(subBoard))
            {
                return 0;
            }
            sum += EvalMetaRow(subBoard[0],subBoard[1],subBoard[2]);
            sum += EvalMetaRow(subBoard[3],subBoard[4],subBoard[5]);
            sum += EvalMetaRow(subBoard[6],subBoard[7],subBoard[8]);
            sum += EvalMetaRow(subBoard[0],subBoard[3],subBoard[6]);
            sum += EvalMetaRow(subBoard[1],subBoard[4],subBoard[7]);
            sum += EvalMetaRow(subBoard[2],subBoard[5],subBoard[8]);
            sum += EvalMetaRow(subBoard[0],subBoard[4],subBoard[8]);
            sum += EvalMetaRow(subBoard[2],subBoard[4],subBoard[6]);
            break;
        case  X:
            sum = 21;
            break;
        case O:
            sum = -21;
            break;
    }
    return sum;
}

double EvalFullBoard(int* board){
    int i;
    double metaBoard[9];
    int winner = BoardWinner(board);
    switch(winner)
    {
        case BLANK:
            if(IsBoardFull(board))
            {
                return 0;
            }
            for(i = 0; i < 9; i++)
            {
                metaBoard[i] = EvalSubBoard(board + 9*i);
            }
            return EvalMetaBoard(metaBoard);
        case X:
            return (double)21;
        case O:
            return (double)-21;
    }
    return 98;
}

__device__ int EvalFullBoardKenel(int* board){
    switch(BoardWinner(board))
    {
        case X:
            return 1;
        case O:
            return -1;
        case BLANK:
            return 0;
    }
    return 0;
}

__device__ __host__ int PlaceMoveinSubBoard(int* board, int lastMove, int placement, int mark){
    int subBoard, freeSquares, i;
    subBoard = lastMove % 9;
    freeSquares = 0;
    for(i = 0; i < 9; i++)
    {
        if(board[subBoard* 9 + i] == 0)
        {
            if(freeSquares == placement)
            {
                board[subBoard* 9 + i] = mark;
                freeSquares = i;
                break;
            }
            freeSquares++;
        }
    }
    if( SubBoardWinner(board + subBoard * 9 ) != 0 )
    {
        for(i = 0; i < 9; i++)
        {
            board[subBoard* 9 + i] = mark;
        }
    }
    return subBoard * 9 + freeSquares;
}

__device__ __host__ int PlaceMarkinNthFree(int* board, int lastMove, int placement, int mark){
    int subBoard, freeSquares, i;
    freeSquares = 0;
    for(i = 0; i < 81; i++)
    {
        if(board[i] == 0)
        {

            if(freeSquares == placement)
            {
                board[i] = mark;
                freeSquares = i;
                break;
            }
            freeSquares++;
        }
    }
    subBoard = i / 9;
    if( SubBoardWinner(board + subBoard * 9 ) != 0 )
    {
        for(i = 0; i < 9; i++)
        {
            board[subBoard* 9 + i] = mark;
        }
    }
    return subBoard * 9 + freeSquares;
}

int playRandomMove(int* board, int lastMove, int mark){

    int subBoard = lastMove%9;
    bool fullBoard = SubBoardWinner(board+9*subBoard) != 0 || IsSubBoardFull(board+subBoard*9) ;
    int numOfMoves = NumberOfPossibleMoves(board, lastMove, fullBoard);
    int index = rand() % (numOfMoves);
    if(fullBoard)
    {
        return PlaceMarkinNthFree(board, lastMove, index, mark);
    }
    else
    {
       return PlaceMoveinSubBoard(board, lastMove, index, mark);
    }
}

__device__ int playRandomMove(int* board, int lastMove, int mark, hiprandState_t  state){

    int subBoard = lastMove%9;
    bool fullBoard = SubBoardWinner(board+9*subBoard) != 0 || IsSubBoardFull(board+subBoard*9) ;
    int numOfMoves = NumberOfPossibleMoves(board, lastMove, fullBoard);
    int index = hiprand(&state) % (numOfMoves);
    if(fullBoard)
    {
        return PlaceMarkinNthFree(board, lastMove, index, mark);
    }
    else
    {
       return PlaceMoveinSubBoard(board, lastMove, index, mark);
    }
}


int MonteCarlo(int* board, int lastMove, int mark, int numRuns){

    int fakeBoard[81];
    int fakeLastMove;
    int fakeMark;
    int subBoard = lastMove%9;
    bool fullBoard = SubBoardWinner(board+9*subBoard) != 0 || IsSubBoardFull(board+subBoard*9) ;
    int numOfMoves = NumberOfPossibleMoves(board, lastMove, fullBoard);
    double score [70];
    for(int i = 0; i < 70; i++)
    {
        score[i] = 0;
    }
    for(int i = 0; i < numRuns; i++)
    {
        for(int j = 0; j < 81; j++)
        {

            fakeBoard[j] = board[j];
              fakeLastMove = lastMove;
          }
        int index = i % (numOfMoves);
        fakeMark = mark;
        if(BoardWinner(fakeBoard) == 0 && !IsBoardFull(fakeBoard)){
            if(fullBoard)
            {
                fakeLastMove = PlaceMarkinNthFree(fakeBoard, fakeLastMove, index, fakeMark);
            }
            else
            {
               fakeLastMove = PlaceMoveinSubBoard(fakeBoard, fakeLastMove, index, fakeMark);
            }
            fakeMark = fakeMark * -1;
            while(BoardWinner(fakeBoard) == 0 && !IsBoardFull(fakeBoard))
            {
                fakeLastMove = playRandomMove(fakeBoard, fakeLastMove, fakeMark);
                fakeMark = -1 * fakeMark;
            }
        }
        score[i % numOfMoves] = EvalFullBoard(fakeBoard) + score[i % numOfMoves];
    }
    int winningIndex = 0;
    if(mark == X)
    {
        double max = score[0];
        for(int i = 0; i < numOfMoves; i++)
        {
            if(score[i] > max)
            {
                winningIndex = i;
                max = score[i];
            }
        }
    }
    else
    {
        double min = score[0];
        for(int i = 0; i < numOfMoves; i++)
        {
            if(score[i] < min)
            {
                winningIndex = i;
                min = score[i];


            }
        }
    }

    if(fullBoard)
    {
        return PlaceMarkinNthFree(board, lastMove, winningIndex, mark);
    }
    else
    {
       return PlaceMoveinSubBoard(board, lastMove, winningIndex, mark);
    }
}

__global__ void MonteCarloKernel(int* board, int* lastMove, int* mark, bool* fullBoard, int* numOfMoves, int* score, int Runs){
    extern  __shared__ int shared[];
    int tId = threadIdx.x + (blockIdx.x * blockDim.x);
    int thread = threadIdx.x;
    hiprandState_t  state;
    hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);
    int o_board[81];
    int fakeBoard[81];
    int fakeLastMove;
    int fakeMark;
    if(thread < *numOfMoves)
    {
        shared[thread] = 0;
    }
    for(int j = 0; j < 81; j++)
    {
        o_board[j] = board[j];
    }
    __syncthreads();
    //offset by tID to reduce collisions on the scores
    for(int i = 0+tId; i < Runs +tId; i++)
    {
        //reset the board in local mem
        for(int j = 0; j < 81; j++)
        {
            fakeBoard[j] = o_board[j];
        }
        int index = i % (*numOfMoves);
        fakeMark = *mark;
        fakeLastMove = *lastMove;
        if(BoardWinner(fakeBoard) == 0 && !IsBoardFull(fakeBoard)){
            if(*fullBoard)
            {
                fakeLastMove = PlaceMarkinNthFree(fakeBoard, fakeLastMove, index, fakeMark);
            }
            else
            {
               fakeLastMove = PlaceMoveinSubBoard(fakeBoard, fakeLastMove, index, fakeMark);
            }
            fakeMark = fakeMark * -1;
            while(BoardWinner(fakeBoard) == 0 && !IsBoardFull(fakeBoard))
            {
                fakeLastMove = playRandomMove(fakeBoard, fakeLastMove, fakeMark, state);
                fakeMark = -1 * fakeMark;
            }
        }
        atomicAdd(&shared[i%(*numOfMoves)], EvalFullBoardKenel(fakeBoard));
    }
    __syncthreads();
    if(thread < *numOfMoves)
    {
        atomicAdd(&score[thread], shared[thread]);
    }
}


int ParMonteCarlo(int* board, int lastMove, int mark, int Runs)
{
    int *d_board, *d_score ,*d_numOfMoves, *d_mark, *d_lastMove;
    bool *d_fullBoard;
    int score[70];
    memset(score, 0, sizeof(int) * 70);
    int subBoard = lastMove%9;
    bool fullBoard = SubBoardWinner(board+9*subBoard) != 0 || IsSubBoardFull(board+subBoard*9);
    int numOfMoves = NumberOfPossibleMoves(board, lastMove, fullBoard);
    hipMalloc(&d_board, sizeof(int) * 81);
    hipMalloc(&d_score, sizeof(int) * 70);
    hipMalloc(&d_mark ,sizeof(int));
    hipMalloc(&d_lastMove ,sizeof(int));
    hipMalloc(&d_numOfMoves ,sizeof(int));
    hipMalloc(&d_fullBoard ,sizeof(bool));
    hipMemset(d_score, 0, sizeof(int) * 70);
    hipMemcpy(d_board, board, sizeof(int) *81, hipMemcpyHostToDevice);
    hipMemcpy(d_mark,&mark,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_numOfMoves,&numOfMoves,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_lastMove,&lastMove,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_fullBoard,&fullBoard,sizeof(bool),hipMemcpyHostToDevice);
    MonteCarloKernel<<<NUM_BLOCKS,NUM_THREAD, sizeof(int) * 70>>>(d_board, d_lastMove, d_mark, d_fullBoard, d_numOfMoves, d_score, Runs);
    hipDeviceSynchronize();
    hipMemcpy(score, d_score, sizeof(int)*70, hipMemcpyDeviceToHost);
    int winningIndex = 0;
    if(mark == X)
    {
        double max = score[0];
        for(int i = 0; i < 70; i++)
        {
            if(score[i] > max)
            {
                winningIndex = i;
                max = score[i];
            }
        }
    }
    else
    {
        double min = score[0];
        for(int i = 0; i < 70; i++)
        {
            if(score[i] < min)
            {
                winningIndex = i;
                min = score[i];
            }
        }
    }
    hipFree(d_board);
    hipFree(d_score);
    hipFree(d_mark);
    hipFree(d_numOfMoves);
    hipFree(d_lastMove);
    hipFree(d_fullBoard);
    if(fullBoard)
    {
        return PlaceMarkinNthFree(board, lastMove, winningIndex, mark);
    }
    else
    {
       return PlaceMoveinSubBoard(board, lastMove, winningIndex, mark);
    }
}


int main()
{
    clock_t start;
    clock_t diff;
    clock_t end;
    clock_t ParTime = 0;
    clock_t SeqTime = 0;
    int Xwin = 0;
    int Ywin = 0;
    srand(time(NULL));
    for(int i = 0; i < 10; i++)
    {
        ParTime = 0;
        SeqTime = 0;
        int board[81];
        memset(board, BLANK, sizeof(int)*81);
        int lastMove = 0;
        int mark = 1;
        bool test= true;
        while(BoardWinner(board) == 0 && !IsBoardFull(board) )
        {
            if(test)
            {
                printf("Monte Carlo Turn in Parallel\n");
                start = clock();
                lastMove = ParMonteCarlo(board, lastMove, mark, RUNS_PER_THREAD);
                end = clock();
                diff =end -start;
                ParTime += diff;

                printf("Par Time: %d\n", diff);
            }
            else
            {
                printf("Monte Carlo Turn in Sequence\n");
                start = clock();
                lastMove = MonteCarlo(board, lastMove, mark, NUM_SEQ_LOOPS);
                end = clock();
                diff = end-start;
                SeqTime += diff;
                printf("Seq Time: %d\n", diff);
            }
            mark = mark * -1;
            test = !test;
            PrintBoard(board);

        }
        if(BoardWinner(board) == X)
        {
            Xwin++;
        }
        else if (BoardWinner(board)== O)
        {
            Ywin++;
        }
        printf("Parallel Time Total %d, Seq Time Total: %d\n",ParTime, SeqTime );
        printf("BoardWinner: ");
        printSquare(BoardWinner(board));
        printf("\n");
    }
    printf("X won %d times\n out of 10\n",Xwin);
    printf("O won %d times\n out of 10\n",Ywin);
    return 0;

}
